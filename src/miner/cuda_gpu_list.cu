#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <sstream>
#include <algorithm>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "cuda_gpu_list.h"

/*
typedef struct {
	int device_id;
	const char *device_name;
	int device_arch[2];
	int device_mpcount;
	int device_blocks;
	int device_threads;
	int device_bfactor;
	int device_bsleep;
	int syncMode;

	uint32_t *d_input;
	uint32_t inputlen;
	uint32_t *d_result_count;
	uint32_t *d_result_nonce;
	uint32_t *d_long_state;
	uint32_t *d_ctx_state;
	uint32_t *d_ctx_a;
	uint32_t *d_ctx_b;
	uint32_t *d_ctx_key1;
	uint32_t *d_ctx_key2;
	uint32_t *d_ctx_text;
	std::string name;
	size_t free_device_memory;
	size_t total_device_memory;
} nvid_ctx;
*/
/** execute and check a CUDA api command
*
* @param id gpu id (thread id)
* @param msg message string which should be added to the error message
* @param ... CUDA api command
*/
#define CUDA_CHECK_MSG(id, msg, ...) { \
	hipError_t error = __VA_ARGS__; \
	if(error!=hipSuccess){	\
		std::cerr << "[CUDA] Error gpu " << id << ": <" << __FILE__ << ">:" << __LINE__; \
		std::cerr << msg << std::endl;                                         \
		throw std::runtime_error(std::string("[CUDA] Error: ") + std::string(hipGetErrorString(error))); \
	} \
} \
( (void) 0 )

/** execute and check a CUDA api command
*
* @param id gpu id (thread id)
* @param ... CUDA api command
*/
#define CUDA_CHECK(id, ...) CUDA_CHECK_MSG(id, "", __VA_ARGS__)

extern "C" int cuda_get_devicecount(int* deviceCount)
{
	hipError_t err;
	*deviceCount = 0;
	err = hipGetDeviceCount(deviceCount);
	if (err != hipSuccess)
	{
		if (err == hipErrorNoDevice)
			printf("ERROR: NVIDIA no CUDA device found!\n");
		else if (err == hipErrorInsufficientDriver)
			printf("WARNING: NVIDIA Insufficient driver!\n");
		else
			printf("WARNING: NVIDIA Unable to query number of CUDA devices!\n");
		return 0;
	}

	return 1;
}

extern "C" int cuda_get_deviceinfo(nvid_ctx* ctx)
{
	hipError_t err;
	int version;

	err = hipDriverGetVersion(&version);
	if (err != hipSuccess)
	{
		printf("Unable to query CUDA driver version! Is an nVidia driver installed?\n");
		return 1;
	}

	if (version < CUDART_VERSION)
	{
		printf("Driver does not support CUDA %d.%d API! Update your nVidia driver!\n", CUDART_VERSION / 1000, (CUDART_VERSION % 1000) / 10);
		return 1;
	}

	int GPU_N;
	if (cuda_get_devicecount(&GPU_N) == 0)
	{
		return 1;
	}

	if (ctx->device_id >= GPU_N)
	{
		printf("Invalid device ID!\n");
		return 1;
	}

	hipDeviceProp_t props;
	err = hipGetDeviceProperties(&props, ctx->device_id);
	if (err != hipSuccess)
	{
		printf("\nGPU %d: %s\n%s line %d\n", ctx->device_id, hipGetErrorString(err), __FILE__, __LINE__);
		return 1;
	}

	ctx->device_name = strdup(props.name);
	ctx->device_mpcount = props.multiProcessorCount;
	ctx->device_arch[0] = props.major;
	ctx->device_arch[1] = props.minor;

	const int gpuArch = ctx->device_arch[0] * 10 + ctx->device_arch[1];

	//ctx->name = std::string(strdup(props.name));

	std::vector<int> arch;
#define XMRSTAK_PP_TOSTRING1(str) #str
#define XMRSTAK_PP_TOSTRING(str) XMRSTAK_PP_TOSTRING1(str)
	//char const * archStringList = XMRSTAK_PP_TOSTRING(XMRSTAK_CUDA_ARCH_LIST);
	char const * archStringList = "50"; // list generated in cmakelists file
#undef XMRSTAK_PP_TOSTRING
#undef XMRSTAK_PP_TOSTRING1
	std::stringstream ss(archStringList);

	//transform string list sperated with `+` into a vector of integers
	int tmpArch;
	while (ss >> tmpArch)
		arch.push_back(tmpArch);

	if (gpuArch >= 20 && gpuArch < 30)
	{
		// compiled binary must support sm_20 for fermi
		std::vector<int>::iterator it = std::find(arch.begin(), arch.end(), 20);
		if (it == arch.end())
		{
			printf("WARNING: NVIDIA GPU %d: miner not compiled for the gpu architecture %d.\n", ctx->device_id, gpuArch);
			return 5;
		}
	}
	if (gpuArch >= 30)
	{
		// search the minimum architecture greater than sm_20
		int minSupportedArch = 0;
		/* - for newer architecture than fermi we need at least sm_30
		* or a architecture >= gpuArch
		* - it is not possible to use a gpu with a architecture >= 30
		*   with a sm_20 only compiled binary
		*/
		for (int i = 0; i < arch.size(); ++i)
			if (minSupportedArch == 0 || (arch[i] >= 30 && arch[i] < minSupportedArch))
				minSupportedArch = arch[i];
		if (minSupportedArch < 30 || gpuArch < minSupportedArch)
		{
			printf("WARNING: NVIDIA GPU %d: miner not compiled for the gpu architecture %d.\n", ctx->device_id, gpuArch);
			return 5;
		}
	}

	// set all evice option those marked as auto (-1) to a valid value
	if (ctx->device_blocks == -1)
	{
		/* good values based of my experience
		*	 - 3 * SMX count >=sm_30
		*   - 2 * SMX count for <sm_30
		*/
		ctx->device_blocks = props.multiProcessorCount *
			(props.major < 3 ? 2 : 3);

		// increase bfactor for low end devices to avoid that the miner is killed by the OS
		if (props.multiProcessorCount <= 6)
			ctx->device_bfactor += 2;
	}
	if (ctx->device_threads == -1)
	{
		/* sm_20 devices can only run 512 threads per cuda block
		* `cryptonight_core_gpu_phase1` and `cryptonight_core_gpu_phase3` starts
		* `8 * ctx->device_threads` threads per block
		*/
		ctx->device_threads = 64;
		constexpr size_t byteToMiB = 1024u * 1024u;

		// no limit by default 1TiB
		size_t maxMemUsage = byteToMiB * byteToMiB;
		if (props.major == 6)
		{
			if (props.multiProcessorCount < 15)
			{
				// limit memory usage for GPUs for pascal < GTX1070
				maxMemUsage = size_t(2048u) * byteToMiB;
			}
			else if (props.multiProcessorCount <= 20)
			{
				// limit memory usage for GPUs for pascal GTX1070, GTX1080
				maxMemUsage = size_t(4096u) * byteToMiB;
			}
		}
		if (props.major < 6)
		{
			// limit memory usage for GPUs before pascal
			maxMemUsage = size_t(2048u) * byteToMiB;
		}
		if (props.major == 2)
		{
			// limit memory usage for sm 20 GPUs
			maxMemUsage = size_t(1024u) * byteToMiB;
		}

		if (props.multiProcessorCount <= 6)
		{
			// limit memory usage for low end devices to reduce the number of threads
			maxMemUsage = size_t(1024u) * byteToMiB;
		}

		int* tmp;
		hipError_t err;
		// a device must be selected to get the right memory usage later on
		err = hipSetDevice(ctx->device_id);
		if (err != hipSuccess)
		{
			printf("WARNING: NVIDIA GPU %d: cannot be selected.\n", ctx->device_id);
			return 2;
		}
		// trigger that a context on the gpu will be allocated
		err = hipMalloc(&tmp, 256);
		if (err != hipSuccess)
		{
			printf("WARNING: NVIDIA GPU %d: context cannot be created.\n", ctx->device_id);
			return 3;
		}


		size_t freeMemory = 0;
		size_t totalMemory = 0;
		CUDA_CHECK(ctx->device_id, hipMemGetInfo(&freeMemory, &totalMemory));

		CUDA_CHECK(ctx->device_id, hipFree(tmp));
		// delete created context on the gpu
		CUDA_CHECK(ctx->device_id, hipDeviceReset());

		ctx->total_device_memory = totalMemory;
		ctx->free_device_memory = freeMemory;

		size_t hashMemSize;
		hashMemSize = 2097152llu;

#ifdef WIN32
		/* We use in windows bfactor (split slow kernel into smaller parts) to avoid
		* that windows is killing long running kernel.
		* In the case there is already memory used on the gpu than we
		* assume that other application are running between the split kernel,
		* this can result into TLB memory flushes and can strongly reduce the performance
		* and the result can be that windows is killing the miner.
		* Be reducing maxMemUsage we try to avoid this effect.
		*/
		size_t usedMem = totalMemory - freeMemory;
		if (usedMem >= maxMemUsage)
		{
			printf("WARNING: NVIDIA GPU %d: already %s MiB memory in use, skip GPU.\n",
				ctx->device_id,
				std::to_string(usedMem / byteToMiB).c_str());
			return 4;
		}
		else
			maxMemUsage -= usedMem;

#endif
		// keep 128MiB memory free (value is randomly chosen)
		// 200byte are meta data memory (result nonce, ...)
		size_t availableMem = freeMemory - (128u * byteToMiB) - 200u;
		size_t limitedMemory = std::min(availableMem, maxMemUsage);
		// up to 16kibyte extra memory is used per thread for some kernel (lmem/local memory)
		// 680bytes are extra meta data memory per hash
		size_t perThread = hashMemSize + 16192u + 680u;
		size_t max_intensity = limitedMemory / perThread;
		ctx->device_threads = max_intensity / ctx->device_blocks;
		// use only odd number of threads
		ctx->device_threads = ctx->device_threads & 0xFFFFFFFE;

		if (props.major == 2 && ctx->device_threads > 64)
		{
			// Fermi gpus only support 512 threads per block (we need start 4 * configured threads)
			ctx->device_threads = 64;
		}

	}

	return 0;
}
